#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <signal.h>
#include <string.h>
#include <unistd.h>
#include <errno.h>
#include <time.h>
#include <math.h>

extern "C" {
#include "config.h"
#include "common.h"
#include "logger.h"
#include "arena-allocator-api.h"
#include "mmio.h"
#include "prof-timer.h"
#include "profiling.h"
}

#include "csr-matrix.h"
#include "cuda-timer.h"


LoggerHandler_t hlogger;
ArenaAllocatorHandler_t harena;
CsrMatrix_t mat;
ProfilingData prof_data;

/*!
 * \brief Exit point of the program
 *
 * \details This function is used to deinitialize everything before the program
 *      ends
 *
 * \param exit_code The exit code value
 * \param fmt String that will be formatted and printed
 * \params ... Additional parameters needed to format the string
 */
void panic(int exit_code, const char *fmt, ...) {
    arena_allocator_api_free(&harena);

    if (fmt != NULL) {
        va_list args;
        va_start(args, fmt);
        // Print message to stdout if the exit code is 0, stderr otherwise
        FILE *out = exit_code == EXIT_SUCCESS ? stdout : stderr;
        vfprintf(out, fmt, args);
        va_end(args);
    }
    exit(exit_code);
}

/*!
 * \brief Interrupt signal handler
 *
 * \param signo Signal number
 * \param info Additional info about the signal
 * \param ucontext Signal context information
 */
void int_handler(int signo, siginfo_t *info, void *ucontext) {
    UNUSED(signo);
    UNUSED(info);
    UNUSED(ucontext);
    panic(EXIT_FAILURE, NULL);
}

/*!
 * \brief Print usage message and exit
 *
 * \param argc Number of given arguments
 * \param argv List of arguments
 */
void print_usage_and_exit(int argc, char *argv[]) {
    UNUSED(argc);
    panic(EXIT_SUCCESS, "usage: %s [file.mtx]\n", argv[0]);
}

void setup(void) {
    profiling_init(&prof_data);

    ProfTimerHandler_t htimer;
    prof_timer_init(&htimer);
    prof_timer_start(&htimer);

    logger_init(&hlogger, DEFAULT_LOG_LEVEL, LOGGER_COLORS_ENABLE);
    logger_info(&hlogger, "initializing...\n", "");

    arena_allocator_api_init(&harena);
    csr_matrix_init(&mat);

    prof_timer_stop(&htimer);
    prof_data.tsetup = prof_timer_elapsed(&htimer);
}

/*!
 * \brief Parse a Matrix Market file to get the matrix data
 *
 * \param path The path of the file to parse
 */
void parse_matrix_from_file(char *path) {
    ProfTimerHandler_t htimer;
    prof_timer_init(&htimer);

    ProfTimerHandler_t htim_parse;
    prof_timer_init(&htim_parse);

    prof_timer_start(&htimer);

    logger_info(&hlogger, "parsing file %s\n", path);

    // Open file
    FILE *fp = fopen(path, "r");
    if (fp == NULL) {
        panic(EXIT_FAILURE, strerror(errno));
    }

    // Parse initial banner
    MM_typecode matcode;
    const char *err_msg = "error while processing Matrix Market banner from file\n";
    if (mm_read_banner(fp, &matcode) != 0) {
        fclose(fp);
        panic(EXIT_FAILURE, err_msg);
    }

    // Check supported matrix types
    if (!mm_is_real(matcode) && !mm_is_pattern(matcode) && !mm_is_integer(matcode)) {
        logger_error(&hlogger, "matrix data type not supported\n", "");
        fclose(fp);
        panic(EXIT_FAILURE, err_msg);
    }
    if (!mm_is_general(matcode) && !mm_is_symmetric(matcode)) {
        logger_error(&hlogger, "matrix storage scheme not supported\n", "");
        fclose(fp);
        panic(EXIT_FAILURE, err_msg);
    }

    mat.symmetric = mm_is_symmetric(matcode);

    char *typestr = mm_typecode_to_str(matcode);
    logger_debug(&hlogger, "[%s] %s\n", matcode, typestr);
    free(typestr);

    // Get matrix info
    err_msg = "error while processing Matrix Market data from file\n";
    int row_count, col_count, nz;
    if (mm_read_mtx_crd_size(fp, &row_count, &col_count, &nz) != 0) {
        logger_error(&hlogger, "could not process Matrix Market coordinate size\n", "");
        fclose(fp);
        panic(EXIT_FAILURE, err_msg);
    }
    mat.row_count = row_count;
    mat.col_count = col_count;
    mat.nz = nz;

    const char *info_fmt = "\n\n    +---------- MATRIX INFO ----------+\n"
                           "    |                                 |\n"
                           "    |   o Symmetric: %14s   |\n"
                           "    |   o Rows: %19d   |\n"
                           "    |   o Columns: %16d   |\n"
                           "    |   o Non-zeros: %14d   |\n"
                           "    \\_________________________________/\n\n";
    logger_info(&hlogger, info_fmt, mat.symmetric ? "Yes" : "No", mat.row_count, mat.col_count, mat.nz);
    if (mat.nz > LARGE_MATRIX_NZ_THRESHOLD)
        logger_warning(&hlogger, "loading matrix with a large number of non-zeros!!!\n", "");

    // Allocate memory for the matrix data
    prof_timer_start(&htim_parse);

    logger_info(&hlogger, "allocating memory for the matrix...\n", "");
    mat.rows = (dsize_t *)arena_allocator_api_calloc(&harena, sizeof(*mat.rows), mat.nz);
    mat.cols = (dsize_t *)arena_allocator_api_calloc(&harena, sizeof(*mat.cols), mat.nz);
    mat.data = (dtype_t *)arena_allocator_api_calloc(&harena, sizeof(*mat.data), mat.nz);
    if (mat.rows == NULL || mat.cols == NULL || mat.data == NULL) {
        logger_error(&hlogger, "could not allocate enough memory for the matrix data\n", "");
        fclose(fp);
        panic(EXIT_FAILURE, strerror(errno));
    }
    memset(mat.rows, 0, mat.nz * sizeof(*mat.rows));
    memset(mat.cols, 0, mat.nz * sizeof(*mat.cols));
    memset(mat.data, 0, mat.nz * sizeof(*mat.data));

    prof_timer_stop(&htim_parse);
    prof_data.tparse.allocation = prof_timer_elapsed(&htim_parse);

    // Parse matrix data from file line by line
    logger_info(&hlogger, "parsing Matrix Market file data...\n", "");
    for (dsize_t i = 0; i < mat.nz; ++i) {
        int r, c;
        double real = 1, imm = 1;

        prof_timer_start(&htim_parse);

        if (mm_read_mtx_crd_entry(fp, &r, &c, &real, &imm, matcode) != 0) {
            logger_error(&hlogger, "could not parse Matrix Market data\n", "");
            fclose(fp);
            panic(EXIT_FAILURE, err_msg);
        }

        prof_timer_stop(&htim_parse);
        prof_data.tparse.io += prof_timer_elapsed(&htim_parse);

        if (i % MAX(1, (mat.nz / 10)) == 0) {
            logger_debug(&hlogger, "progress %.0f%%\n", (float)i / mat.nz * 100.f);
        }

        /*! Rows and columns indices starts from 1 */
        mat.rows[i] = r - 1;
        mat.cols[i] = c - 1;
        mat.data[i] = real;

        /*
         * Update floating point operation count
         * The operation are a multiplication and an addition for each non-zero
         * Symmetric matrices has to be taken in account
         */
        prof_data.flop += 2U;
        if (csr_is_symmetric(&mat))
            prof_data.flop += 2U;
    }

    fclose(fp);

    prof_timer_stop(&htimer);
    prof_data.tparse.total = prof_timer_elapsed(&htimer);

    logger_debug(&hlogger, "parsing done!!!\n", "");
}

void construct_csr_matrix(void) {
    ProfTimerHandler_t htimer;
    prof_timer_init(&htimer);

    ProfTimerHandler_t htim_csr;
    prof_timer_init(&htim_csr);

    prof_timer_start(&htimer);

    logger_info(&hlogger, "constructing CSR matrix...\n", "");

    prof_timer_start(&htim_csr);

    // Sort rows
    csr_sort(&mat);

    prof_timer_stop(&htim_csr);
    prof_data.tcsr.sort = prof_timer_elapsed(&htim_csr);

    logger_info(&hlogger, "generating rows prefix sum...\n", "");

    prof_timer_start(&htim_csr);

    dsize_t *rows = mat.rows;
    mat.rows = (dsize_t *)arena_allocator_api_calloc(&harena, sizeof(*mat.rows), mat.row_count + 1);
    memset(mat.rows, 0, sizeof(*mat.rows) * (mat.row_count + 1));

    // Pack matrix rows
    csr_pack(&mat, rows);

    prof_timer_stop(&htim_csr);
    prof_data.tcsr.pack = prof_timer_elapsed(&htim_csr);

    prof_timer_stop(&htimer);
    prof_data.tcsr.total = prof_timer_elapsed(&htimer);
}

dtype_t *generate_input_vector(dsize_t count) {
    ProfTimerHandler_t htimer;
    prof_timer_init(&htimer);
    prof_timer_start(&htimer);

    logger_info(&hlogger, "generating input vector...\n", "");
    dtype_t *x = (dtype_t *)arena_allocator_api_calloc(&harena, sizeof(*x), count);
    for (dsize_t i = 0; i < count; ++i) {
        x[i] = (rand() % RAND_MAX) / 1e6;
    }

    prof_timer_stop(&htimer);
    prof_data.tgen = prof_timer_elapsed(&htimer);
    return x;
}

__global__ void spmv_mul(CsrMatrix_t *mat, dtype_t *x) {
    const dsize_t batch_size = gridDim.x; 
    const dsize_t stride = blockDim.x;
    const dsize_t block = blockIdx.x;
    const dsize_t idx = threadIdx.x;

    UNUSED(batch_size);

    const dsize_t i = block * stride + idx;
    if (i >= mat->nz)
        return;

    /* Binary search row */
    dsize_t j = 0;
    dsize_t count = mat->row_count + 1;
    while (count > 0) {
        count /= 2;
        dsize_t m = j + count;
        if (mat->rows[m] < i) {
            j = m + 1;
        }
    }

    /* Multiply values */
    dsize_t r = mat->rows[j];
    dsize_t c = i - r;
    mat->data[i] *= x[c];
    if (mat->symmetric && r != c)
        mat->data[i] *= x[r];
}

__global__ void spmv_add(CsrMatrix_t *mat) {
    dsize_t r = blockIdx.y;
    dsize_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    dsize_t lane = threadIdx.x % MAX_THREAD_PER_WARP_COUNT;
    dsize_t col_count = mat->rows[r + 1] - mat->rows[r];
    if (idx >= col_count)
        return;
    dsize_t j = mat->rows[r] + idx;
    dtype_t val = mat->data[j];

    for (dsize_t off = MAX_THREAD_PER_WARP_COUNT / 2; off > 0; off >>= 1) {
        val += __shfl_down_sync(0xffffffff, val, off);
    }

    if (lane == 0)
        mat->data[j] = val;
}

void output_dump(char filename[128], dtype_t *y, dsize_t count);
dtype_t *dispatch(CsrMatrix_t *mat, dtype_t *x) {
    ProfTimerHandler_t htimer;
    prof_timer_init(&htimer);

    CudaTimerHandler_t htim_spmv;
    cuda_timer_init(&htim_spmv);

    prof_timer_start(&htimer);

    logger_info(&hlogger, "calculating sparse matrix vector product...\n", "");

    ProfTimerHandler_t htim_alloc;
    prof_timer_init(&htim_alloc);
    prof_timer_start(&htim_alloc);

    CsrMatrix_t *d_mat;
    dsize_t *d_rows, *d_cols;
    dtype_t *d_data, *d_x;
    hipMallocManaged(&d_mat, sizeof(*d_mat));
    hipMalloc(&d_rows, (mat->row_count + 1) * sizeof(*d_rows));
    hipMalloc(&d_cols, mat->nz * sizeof(*d_cols));
    hipMalloc(&d_data, mat->nz * sizeof(*d_data));
    hipMalloc(&d_x, mat->col_count * sizeof(*d_x));

    hipMemcpy(d_mat, mat, sizeof(*d_mat), hipMemcpyHostToDevice);
    hipMemcpy(d_rows, mat->rows, (mat->row_count + 1) * sizeof(*d_rows), hipMemcpyHostToDevice);
    hipMemcpy(d_cols, mat->cols, mat->nz * sizeof(*d_cols), hipMemcpyHostToDevice);
    hipMemcpy(d_data, mat->data, mat->nz * sizeof(*d_data), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, mat->col_count * sizeof(*d_x), hipMemcpyHostToDevice);

    d_mat->rows = d_rows;
    d_mat->cols = d_cols;
    d_mat->data = d_data;

    dtype_t *y = (dtype_t *)arena_allocator_api_calloc(&harena, sizeof(*y), mat->row_count);

    prof_timer_stop(&htim_alloc);
    prof_data.tspmv.allocation = prof_timer_elapsed(&htim_alloc);

    logger_debug(&hlogger, "rows: host %p = device %p [%s]\n", mat->rows, d_rows, mat->rows == d_rows ? "EQUAL" : "NOT EQUAL");
    logger_debug(&hlogger, "cols: host %p = device %p [%s]\n", mat->cols, d_cols, mat->cols == d_cols ? "EQUAL" : "NOT EQUAL");
    logger_debug(&hlogger, "data: host %p = device %p [%s]\n", mat->data, d_data, mat->data == d_data ? "EQUAL" : "NOT EQUAL");

    dsize_t max_col = 0;
    for (dsize_t r = 0; r < mat->row_count; ++r) {
        max_col = MAX(max_col, mat->rows[r + 1] - mat->rows[r]);
    }
    logger_debug(&hlogger, "maximum column length: %lu\n", max_col);

    for (dint_t i = -TSKIP; i < TITER; ++i) {
        memset(y, 0, mat->row_count * sizeof(*y));
        hipMemcpy(d_data, mat->data, mat->nz * sizeof(*d_data), hipMemcpyHostToDevice);

        // Calculate product
        const dsize_t b1 = mat->nz < MAX_THREAD_COUNT ? 1 : MIN(MAX_BLOCK_COUNT, mat->nz / MAX_THREAD_COUNT);
        dsize_t t1 = MIN(MAX_THREAD_COUNT, mat->nz / b1);
        cuda_timer_start(&htim_spmv);
        spmv_mul<<<b1, t1>>>(d_mat, d_x);
        cuda_timer_synchronize(&htim_spmv);
        cuda_timer_stop(&htim_spmv);

        // Calculate sum
        const dim3 b2(max_col / MAX_THREAD_PER_WARP_COUNT, mat->row_count, 0);
        const dsize_t t2 = MAX_THREAD_PER_WARP_COUNT;
        cuda_timer_start(&htim_spmv);
        spmv_add<<<b2, t2>>>(d_mat);
        cuda_timer_synchronize(&htim_spmv);
        cuda_timer_stop(&htim_spmv);

        // Copy result to host
        hipMemcpy(mat->data, d_data, mat->nz * sizeof(*mat->data), hipMemcpyDeviceToHost);
        for (dsize_t r = 0; r < mat->row_count; ++r) {
            dsize_t j = mat->rows[r];
            for (dsize_t c = 0; c < b2.x; ++c) {
                y[r] += mat->data[j + c];
            }
        }

        if (i >= 0) {
            prof_data.tspmv.t[i] = cuda_timer_elapsed(&htim_spmv);
            logger_debug(&hlogger, "iteration %d: %2.5f s\n", i + 1, prof_data.tspmv.t[i]);
        }
        else {
            logger_debug(&hlogger, "warm-up %d: %2.5f s\n", TSKIP + i + 1, cuda_timer_elapsed(&htim_spmv));
        }
    }

    output_dump((char *)"mat", mat->data, mat->nz);

    prof_timer_stop(&htimer);
    prof_data.tspmv.total = prof_timer_elapsed(&htimer);

    cuda_timer_deinit(&htim_spmv);
    hipFree(d_mat);
    hipFree(d_rows);
    hipFree(d_cols);
    hipFree(d_data);
    hipFree(d_x);
    return y;
}

void output_dump(char filename[128], dtype_t *y, dsize_t count) {
    const dsize_t len = 256;
    char path[len];
    memset(path, 0, len * sizeof(*path));
    strncpy(path, filename, 128);

    // Open output file
    const time_t t = time(NULL);
    struct tm *tp = localtime(&t);
    strftime(path + strlen(path), len, "-%F-%T.mtx", tp);

    FILE *fp = fopen(path, "w+");
    if (fp == NULL) {
        logger_error(&hlogger, strerror(errno), "");
        return;
    }

    // Write banner
    MM_typecode matcode;
    mm_set_matrix(&matcode);
    mm_set_array(&matcode);
    mm_set_real(&matcode);
    mm_set_general(&matcode);
    if (mm_write_banner(fp, matcode) != 0) {
        logger_error(&hlogger, "failed to write output banner to file\n", "");
        fclose(fp);
        return;
    }

    // Write size
    if (mm_write_mtx_array_size(fp, count, 1) != 0) {
        logger_error(&hlogger, "failed to write output array size to file\n", "");
        fclose(fp);
        return;
    }

    // Write data
    for (dsize_t i = 0; i < count; ++i) {
        if (fprintf(fp, "%f\n", y[i]) < 0) {
            logger_error(&hlogger, "failed to write output data to file\n", "");
            fclose(fp);
            return;
        }
    }

    fclose(fp);
}

int main(int argc, char *argv[]) {
    {
        /* Setup signal handling */
        struct sigaction act = { 0 };
        act.sa_flags = SA_SIGINFO;
        act.sa_sigaction = &int_handler;
        if (sigaction(SIGINT, &act, NULL) == -1) {
            panic(EXIT_FAILURE, NULL);
        }
    }

    ProfTimerHandler_t htimer;
    prof_timer_init(&htimer);
    prof_timer_start(&htimer);

    /*  1. Check arguments                                                   */
    if (argc != 2) {
        print_usage_and_exit(argc, argv);
    }

    /*  2. Initialize everything                                             */
    setup();

    /*  3. Read matrix from file                                             */
    parse_matrix_from_file(argv[1]);

    /*  4. Construct matrix with CSR format                                  */
    construct_csr_matrix();

    /*  5. Generate random vector                                            */
    dtype_t *x = generate_input_vector(mat.col_count);

    /*  6. Calculate matrix-vector product                                   */
    dtype_t *y = dispatch(&mat, x);

    prof_timer_stop(&htimer);
    prof_data.ttotal = prof_timer_elapsed(&htimer);

    /*  7. Print results                                                     */
    profiling_dump(&prof_data);

#ifdef DUMP_OUTPUT
    const dsize_t len = 128;
    char filename[len];
    memset(filename, 0, len * sizeof(*filename));
    strncpy(filename, "input-dump", len);
    output_dump(filename, x, mat.col_count);
    strncpy(filename, "output-dump", len);
    output_dump(filename, y, mat.row_count);
#else
    UNUSED(y);
#endif // DUMP_OUPUT

    panic(EXIT_SUCCESS, NULL);
    return 0;
}
