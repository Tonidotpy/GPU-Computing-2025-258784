#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <signal.h>
#include <string.h>
#include <unistd.h>
#include <errno.h>
#include <time.h>
#include <math.h>

extern "C" {
#include "config.h"
#include "common.h"
#include "logger.h"
#include "arena-allocator-api.h"
#include "mmio.h"
#include "prof-timer.h"
#include "profiling.h"
}

#include "csr-matrix.h"
#include "cuda-timer.h"

LoggerHandler_t hlogger;
ArenaAllocatorHandler_t harena;
CsrMatrix_t mat;
ProfilingData prof_data;

void output_dump(char filename[128], dtype_t *y, dsize_t count);

/*!
 * \brief Exit point of the program
 *
 * \details This function is used to deinitialize everything before the program
 *      ends
 *
 * \param exit_code The exit code value
 * \param fmt String that will be formatted and printed
 * \params ... Additional parameters needed to format the string
 */
void panic(int exit_code, const char *fmt, ...) {
    arena_allocator_api_free(&harena);

    if (fmt != NULL) {
        va_list args;
        va_start(args, fmt);
        // Print message to stdout if the exit code is 0, stderr otherwise
        FILE *out = exit_code == EXIT_SUCCESS ? stdout : stderr;
        vfprintf(out, fmt, args);
        va_end(args);
    }
    exit(exit_code);
}

/*!
 * \brief Interrupt signal handler
 *
 * \param signo Signal number
 * \param info Additional info about the signal
 * \param ucontext Signal context information
 */
void int_handler(int signo, siginfo_t *info, void *ucontext) {
    UNUSED(signo);
    UNUSED(info);
    UNUSED(ucontext);
    panic(EXIT_FAILURE, NULL);
}

/*!
 * \brief Print usage message and exit
 *
 * \param argc Number of given arguments
 * \param argv List of arguments
 */
void print_usage_and_exit(int argc, char *argv[]) {
    UNUSED(argc);
    panic(EXIT_SUCCESS, "usage: %s [file.mtx]\n", argv[0]);
}

void setup(void) {
    profiling_init(&prof_data);

    ProfTimerHandler_t htimer;
    prof_timer_init(&htimer);
    prof_timer_start(&htimer);

    logger_init(&hlogger, DEFAULT_LOG_LEVEL, LOGGER_COLORS_ENABLE);
    logger_info(&hlogger, "initializing...\n", "");

    arena_allocator_api_init(&harena);
    csr_matrix_init(&mat);

    prof_timer_stop(&htimer);
    prof_data.tsetup = prof_timer_elapsed(&htimer);
}

/*!
 * \brief Parse a Matrix Market file to get the matrix data
 *
 * \param path The path of the file to parse
 */
void parse_matrix_from_file(char *path) {
    ProfTimerHandler_t htimer;
    prof_timer_init(&htimer);

    ProfTimerHandler_t htim_parse;
    prof_timer_init(&htim_parse);

    prof_timer_start(&htimer);

    logger_info(&hlogger, "parsing file %s\n", path);

    // Open file
    FILE *fp = fopen(path, "r");
    if (fp == NULL) {
        panic(EXIT_FAILURE, strerror(errno));
    }

    // Parse initial banner
    MM_typecode matcode;
    const char *err_msg = "error while processing Matrix Market banner from file\n";
    if (mm_read_banner(fp, &matcode) != 0) {
        fclose(fp);
        panic(EXIT_FAILURE, err_msg);
    }

    // Check supported matrix types
    if (!mm_is_real(matcode) && !mm_is_pattern(matcode) && !mm_is_integer(matcode)) {
        logger_error(&hlogger, "matrix data type not supported\n", "");
        fclose(fp);
        panic(EXIT_FAILURE, err_msg);
    }
    if (!mm_is_general(matcode) && !mm_is_symmetric(matcode)) {
        logger_error(&hlogger, "matrix storage scheme not supported\n", "");
        fclose(fp);
        panic(EXIT_FAILURE, err_msg);
    }

    mat.symmetric = mm_is_symmetric(matcode);

    char *typestr = mm_typecode_to_str(matcode);
    logger_debug(&hlogger, "[%s] %s\n", matcode, typestr);
    free(typestr);

    // Get matrix info
    err_msg = "error while processing Matrix Market data from file\n";
    int row_count, col_count, nz;
    if (mm_read_mtx_crd_size(fp, &row_count, &col_count, &nz) != 0) {
        logger_error(&hlogger, "could not process Matrix Market coordinate size\n", "");
        fclose(fp);
        panic(EXIT_FAILURE, err_msg);
    }
    mat.row_count = row_count;
    mat.col_count = col_count;
    mat.nz = nz;

    const char *info_fmt = "\n\n    +---------- MATRIX INFO ----------+\n"
                           "    |                                 |\n"
                           "    |   o Symmetric: %14s   |\n"
                           "    |   o Rows: %19d   |\n"
                           "    |   o Columns: %16d   |\n"
                           "    |   o Non-zeros: %14d   |\n"
                           "    \\_________________________________/\n\n";
    logger_info(&hlogger, info_fmt, mat.symmetric ? "Yes" : "No", mat.row_count, mat.col_count, mat.nz);
    if (mat.nz > LARGE_MATRIX_NZ_THRESHOLD)
        logger_warning(&hlogger, "loading matrix with a large number of non-zeros!!!\n", "");

    // Allocate memory for the matrix data
    prof_timer_start(&htim_parse);

    logger_info(&hlogger, "allocating memory for the matrix...\n", "");
    mat.rows = (dsize_t *)arena_allocator_api_calloc(&harena, sizeof(*mat.rows), mat.nz * 2);
    mat.cols = (dsize_t *)arena_allocator_api_calloc(&harena, sizeof(*mat.cols), mat.nz * 2);
    mat.data = (dtype_t *)arena_allocator_api_calloc(&harena, sizeof(*mat.data), mat.nz * 2);
    if (mat.rows == NULL || mat.cols == NULL || mat.data == NULL) {
        logger_error(&hlogger, "could not allocate enough memory for the matrix data\n", "");
        fclose(fp);
        panic(EXIT_FAILURE, strerror(errno));
    }
    memset(mat.rows, 0, mat.nz * 2 * sizeof(*mat.rows));
    memset(mat.cols, 0, mat.nz * 2 * sizeof(*mat.cols));
    memset(mat.data, 0, mat.nz * 2 * sizeof(*mat.data));

    prof_timer_stop(&htim_parse);
    prof_data.tparse.allocation = prof_timer_elapsed(&htim_parse);

    // Parse matrix data from file line by line
    logger_info(&hlogger, "parsing Matrix Market file data...\n", "");
    for (dsize_t i = 0; i < mat.nz; ++i) {
        int r, c;
        double real = 1, imm = 1;

        prof_timer_start(&htim_parse);

        if (mm_read_mtx_crd_entry(fp, &r, &c, &real, &imm, matcode) != 0) {
            logger_error(&hlogger, "could not parse Matrix Market data\n", "");
            fclose(fp);
            panic(EXIT_FAILURE, err_msg);
        }

        prof_timer_stop(&htim_parse);
        prof_data.tparse.io += prof_timer_elapsed(&htim_parse);

        if (i % MAX(1, (mat.nz / 10)) == 0) {
            logger_debug(&hlogger, "progress %.0f%%\n", (float)i / mat.nz * 100.f);
        }

        /*! Rows and columns indices starts from 1 */
        mat.rows[i] = r - 1;
        mat.cols[i] = c - 1;
        mat.data[i] = real;

        /*
         * Update floating point operation count
         * The operation are a multiplication and an addition for each non-zero
         * Symmetric matrices has to be taken in account
         */
        prof_data.flop += 2U;
        if (csr_is_symmetric(&mat) && r != c) {
            prof_data.flop += 2U;
            ++i;
            ++mat.nz;

            /*! Rows and columns indices starts from 1 */
            mat.rows[i] = c - 1;
            mat.cols[i] = r - 1;
            mat.data[i] = real;
        }
    }

    fclose(fp);

    prof_timer_stop(&htimer);
    prof_data.tparse.total = prof_timer_elapsed(&htimer);

    logger_debug(&hlogger, "parsing done!!!\n", "");
}

void construct_csr_matrix(void) {
    ProfTimerHandler_t htimer;
    prof_timer_init(&htimer);

    ProfTimerHandler_t htim_csr;
    prof_timer_init(&htim_csr);

    prof_timer_start(&htimer);

    logger_info(&hlogger, "constructing CSR matrix...\n", "");

    prof_timer_start(&htim_csr);

    // Sort rows
    csr_sort(&mat);

    prof_timer_stop(&htim_csr);
    prof_data.tcsr.sort = prof_timer_elapsed(&htim_csr);

    logger_info(&hlogger, "generating rows prefix sum...\n", "");

    prof_timer_start(&htim_csr);

    dsize_t *rows = mat.rows;
    mat.rows = (dsize_t *)arena_allocator_api_calloc(&harena, sizeof(*mat.rows), mat.row_count + 1);
    memset(mat.rows, 0, sizeof(*mat.rows) * (mat.row_count + 1));

    // Pack matrix rows
    csr_pack(&mat, rows);

    prof_timer_stop(&htim_csr);
    prof_data.tcsr.pack = prof_timer_elapsed(&htim_csr);

    prof_timer_stop(&htimer);
    prof_data.tcsr.total = prof_timer_elapsed(&htimer);
}

dtype_t *generate_input_vector(dsize_t count) {
    ProfTimerHandler_t htimer;
    prof_timer_init(&htimer);
    prof_timer_start(&htimer);

    logger_info(&hlogger, "generating input vector...\n", "");
    dtype_t *x = (dtype_t *)arena_allocator_api_calloc(&harena, sizeof(*x), count);
    for (dsize_t i = 0; i < count; ++i) {
        x[i] = (rand() % RAND_MAX) / 1e6;
    }

    prof_timer_stop(&htimer);
    prof_data.tgen = prof_timer_elapsed(&htimer);
    return x;
}

__global__ void spmv_mul(CsrMatrix_t *mat, dtype_t *x) {
    const dsize_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ignore indices outside of the matrix bounds
    if (idx >= mat->nz)
        return;
    dsize_t c = mat->cols[idx];
    mat->data[idx] *= x[c];
}

__device__ dsize_t row_binary_search(CsrMatrix_t *mat, dsize_t *rows_idx) {
    dint_t r = 0;
    dint_t end = mat->row_count + 1;
    while (r < end) {
        dsize_t m = r + (end - r) / 2;
        if (rows_idx[m] < blockIdx.x) {
            r = m + 1;
        } else {
            end = m;
        }
    }
    if (r != 0 && blockIdx.x < rows_idx[r])
        --r;
    return r;
}

__global__ void spmv_add(CsrMatrix_t *mat, dsize_t *rows_idx) {
    /* Binary search row of the block */
    dsize_t r = row_binary_search(mat, rows_idx);

    /* Find starting index of the block for the given row (divided by MAX_THREAD_COUNT) */
    dsize_t j = blockIdx.x - rows_idx[r];
    dsize_t k = j * MAX_THREAD_COUNT + threadIdx.x;
    dsize_t nz_per_row = mat->rows[r + 1] - mat->rows[r];

    /* Warp reduction sum */
    if (k >= nz_per_row)
        return;

    /* Create mask to only use threads inside the bounds of the matrix */
    dint_t mask = __ballot_sync(FULL_MASK, k < nz_per_row);
    dtype_t val = mat->data[mat->rows[r] + k];
    for (dsize_t off = MAX_THREAD_PER_WARP_COUNT / 2; off > 0; off >>= 1) {
        val += __shfl_down_sync(mask, val, off);
    }

    dsize_t lane = threadIdx.x % 32;
    if (lane == 0) {
        mat->data[mat->rows[r] + k] = val;
    }
}

dtype_t *dispatch(CsrMatrix_t *mat, dtype_t *x) {
    ProfTimerHandler_t htimer;
    prof_timer_init(&htimer);

    CudaTimerHandler_t htim_spmv;
    cuda_timer_init(&htim_spmv);

    prof_timer_start(&htimer);

    logger_info(&hlogger, "calculating sparse matrix vector product...\n", "");

    ProfTimerHandler_t htim_alloc;
    prof_timer_init(&htim_alloc);
    prof_timer_start(&htim_alloc);

    CsrMatrix_t *d_mat;
    dsize_t *d_rows, *d_cols;
    dtype_t *d_data, *d_x;
    hipMallocManaged(&d_mat, sizeof(*d_mat));
    hipMalloc(&d_rows, (mat->row_count + 1) * sizeof(*d_rows));
    hipMalloc(&d_cols, mat->nz * sizeof(*d_cols));
    hipMalloc(&d_data, mat->nz * sizeof(*d_data));
    hipMalloc(&d_x, mat->col_count * sizeof(*d_x));

    hipMemcpy(d_mat, mat, sizeof(*d_mat), hipMemcpyHostToDevice);
    hipMemcpy(d_rows, mat->rows, (mat->row_count + 1) * sizeof(*d_rows), hipMemcpyHostToDevice);
    hipMemcpy(d_cols, mat->cols, mat->nz * sizeof(*d_cols), hipMemcpyHostToDevice);
    hipMemcpy(d_data, mat->data, mat->nz * sizeof(*d_data), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, mat->col_count * sizeof(*d_x), hipMemcpyHostToDevice);

    d_mat->rows = d_rows;
    d_mat->cols = d_cols;
    d_mat->data = d_data;

    dtype_t *y = (dtype_t *)arena_allocator_api_calloc(&harena, sizeof(*y), mat->row_count);

    prof_timer_stop(&htim_alloc);
    prof_data.tspmv.allocation = prof_timer_elapsed(&htim_alloc);

    logger_debug(&hlogger, "rows: host %p = device %p [%s]\n", mat->rows, d_rows, mat->rows == d_rows ? "EQUAL" : "NOT EQUAL");
    logger_debug(&hlogger, "cols: host %p = device %p [%s]\n", mat->cols, d_cols, mat->cols == d_cols ? "EQUAL" : "NOT EQUAL");
    logger_debug(&hlogger, "data: host %p = device %p [%s]\n", mat->data, d_data, mat->data == d_data ? "EQUAL" : "NOT EQUAL");

    /* Auxiliary arrays needed for the calculation */
    dtype_t *data = (dtype_t *)arena_allocator_api_calloc(&harena, sizeof(*data), mat->nz);
    memcpy(data, mat->data, mat->nz * sizeof(*data));

    dsize_t *rows_idx = (dsize_t *)arena_allocator_api_calloc(&harena, sizeof(*rows_idx), mat->row_count + 1);
    dsize_t *d_rows_idx;
    hipMalloc(&d_rows_idx, (mat->row_count + 1) * sizeof(*d_rows_idx));

    rows_idx[0] = 0;
    for (dsize_t i = 1; i < mat->row_count + 1; ++i) {
        rows_idx[i] = rows_idx[i - 1] + (mat->rows[i] - mat->rows[i - 1]) / MAX_THREAD_COUNT + 1;
    }
    hipMemcpy(d_rows_idx, rows_idx, (mat->row_count + 1) * sizeof(*d_rows_idx), hipMemcpyHostToDevice);

    const dsize_t mul_blocks = mat->nz < MAX_THREAD_COUNT ? 1 : ceil(mat->nz / (double)MAX_THREAD_COUNT);
    const dsize_t mul_tcount = (dsize_t)1 << (dsize_t)ceil(log2(mat->nz / (double)mul_blocks));
    const dsize_t mul_threads_per_block = MIN(MAX_THREAD_COUNT, mul_tcount);
    logger_debug(&hlogger, "Multiplication Blocks/Threads: <%lu, %lu>\n", mul_blocks, mul_threads_per_block);

    dsize_t add_blocks = 0;
    for (dsize_t i = 0; i < mat->row_count; ++i) {
        add_blocks += (dsize_t)ceil((mat->rows[i + 1] - mat->rows[i]) / (double)MAX_THREAD_COUNT);
    }
    const dsize_t add_threads_per_block = MAX_THREAD_COUNT;
    logger_debug(&hlogger, "Addition Blocks/Threads: <%lu, %lu>\n", add_blocks, add_threads_per_block);
    for (dint_t i = -TSKIP; i < TITER; ++i) {
        memset(y, 0, mat->row_count * sizeof(*y));
        hipMemcpy(d_data, data, mat->nz * sizeof(*d_data), hipMemcpyHostToDevice);

        // Calculate product
        cuda_timer_start(&htim_spmv);
        spmv_mul<<<mul_blocks, mul_threads_per_block>>>(d_mat, d_x);
        cuda_timer_synchronize(&htim_spmv);

        // Calculate sum
        spmv_add<<<add_blocks, add_threads_per_block>>>(d_mat, d_rows_idx);
        cuda_timer_synchronize(&htim_spmv);

        // Copy result to host
        hipMemcpy(mat->data, d_data, mat->nz * sizeof(*mat->data), hipMemcpyDeviceToHost);

        // Sum all partial sums
        for (dsize_t r = 0; r < mat->row_count; ++r) {
            for (dsize_t j = mat->rows[r]; j < mat->rows[r + 1]; j += MAX_THREAD_PER_WARP_COUNT) {
                y[r] += mat->data[j];
            }
        }

        cuda_timer_stop(&htim_spmv);

        if (i >= 0) {
            prof_data.tspmv.t[i] = cuda_timer_elapsed(&htim_spmv);
            logger_debug(&hlogger, "iteration %d: %2.5f s\n", i + 1, prof_data.tspmv.t[i]);
        } else {
            logger_debug(&hlogger, "warm-up %d: %2.5f s\n", TSKIP + i + 1, cuda_timer_elapsed(&htim_spmv));
        }
    }

    prof_timer_stop(&htimer);
    prof_data.tspmv.total = prof_timer_elapsed(&htimer);

    cuda_timer_deinit(&htim_spmv);
    hipFree(d_mat);
    hipFree(d_rows);
    hipFree(d_cols);
    hipFree(d_data);
    hipFree(d_x);
    hipFree(d_rows_idx);
    return y;
}

void output_dump(char filename[128], dtype_t *y, dsize_t count) {
    const dsize_t len = 256;
    char path[len];
    memset(path, 0, len * sizeof(*path));
    strncpy(path, filename, 128);

    // Open output file
    const time_t t = time(NULL);
    struct tm *tp = localtime(&t);
    strftime(path + strlen(path), len, "-%F-%T.mtx", tp);

    FILE *fp = fopen(path, "w+");
    if (fp == NULL) {
        logger_error(&hlogger, strerror(errno), "");
        return;
    }

    // Write banner
    MM_typecode matcode;
    mm_set_matrix(&matcode);
    mm_set_array(&matcode);
    mm_set_real(&matcode);
    mm_set_general(&matcode);
    if (mm_write_banner(fp, matcode) != 0) {
        logger_error(&hlogger, "failed to write output banner to file\n", "");
        fclose(fp);
        return;
    }

    // Write size
    if (mm_write_mtx_array_size(fp, count, 1) != 0) {
        logger_error(&hlogger, "failed to write output array size to file\n", "");
        fclose(fp);
        return;
    }

    // Write data
    for (dsize_t i = 0; i < count; ++i) {
        if (fprintf(fp, "%f\n", y[i]) < 0) {
            logger_error(&hlogger, "failed to write output data to file\n", "");
            fclose(fp);
            return;
        }
    }

    fclose(fp);
}

int main(int argc, char *argv[]) {
    {
        /* Setup signal handling */
        struct sigaction act = { 0 };
        act.sa_flags = SA_SIGINFO;
        act.sa_sigaction = &int_handler;
        if (sigaction(SIGINT, &act, NULL) == -1) {
            panic(EXIT_FAILURE, NULL);
        }
    }

    ProfTimerHandler_t htimer;
    prof_timer_init(&htimer);
    prof_timer_start(&htimer);

    /*  1. Check arguments                                                   */
    if (argc != 2) {
        print_usage_and_exit(argc, argv);
    }

    /*  2. Initialize everything                                             */
    setup();

    /*  3. Read matrix from file                                             */
    parse_matrix_from_file(argv[1]);

    /*  4. Construct matrix with CSR format                                  */
    construct_csr_matrix();

    /*  5. Generate random vector                                            */
    dtype_t *x = generate_input_vector(mat.col_count);

    /*  6. Calculate matrix-vector product                                   */
    dtype_t *y = dispatch(&mat, x);

    prof_timer_stop(&htimer);
    prof_data.ttotal = prof_timer_elapsed(&htimer);

    /*  7. Print results                                                     */
    profiling_dump(&prof_data);

#ifdef DUMP_OUTPUT
    const dsize_t len = 128;
    char filename[len];
    memset(filename, 0, len * sizeof(*filename));
    strncpy(filename, "input-dump", len);
    output_dump(filename, x, mat.col_count);
    strncpy(filename, "output-dump", len);
    output_dump(filename, y, mat.row_count);
#else
    UNUSED(y);
#endif // DUMP_OUPUT

    panic(EXIT_SUCCESS, NULL);
    return 0;
}
